
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 4
#define THREADS_PER_BLOCK 256

__host__ void print_vector(const char *name, const int vector[], size_t size) {
    if (name != NULL) {
        printf("%s ", name);
    }
    for (size_t i = 0; i < size; ++i) {
        printf("%2d ", vector[i]);
    }
    printf("\n");
}

__global__ void vector_add_cuda(int result[], const int a[], const int b[],
                                size_t size) {
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < size) {
        result[index] = a[index] + b[index];
    }
}

int main(void) {
    int a[N] = {22, 13, 16, 5};
    int b[N] = {5, 22, 17, 37};
    int c[N];

    int *cuda_a, *cuda_b, *cuda_c;
    hipMalloc(&cuda_a, sizeof a);
    hipMalloc(&cuda_b, sizeof b);
    hipMalloc(&cuda_c, sizeof c);
    hipMemcpy(cuda_a, a, sizeof a, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, sizeof b, hipMemcpyHostToDevice);

    vector_add_cuda<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK,
                      THREADS_PER_BLOCK>>>(cuda_c, cuda_a, cuda_b,
                                           sizeof c / sizeof(int));
    hipDeviceSynchronize();

    hipMemcpy(c, cuda_c, sizeof c, hipMemcpyDeviceToHost);
    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_c);

    print_vector("A", a, sizeof a / sizeof(int));
    print_vector("B", b, sizeof b / sizeof(int));
    print_vector("C", c, sizeof c / sizeof(int));

    return 0;
}
