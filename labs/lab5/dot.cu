
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 4
#define THREADS_PER_BLOCK 256

__host__ void print_vector(const char *name, const int vector[], size_t size) {
    if (name != NULL) {
        printf("%s ", name);
    }
    for (size_t i = 0; i < size; ++i) {
        printf("%2d ", vector[i]);
    }
    printf("\n");
}

__global__ void vector_dot_product_cuda(int *result, const int a[],
                                        const int b[], size_t size) {
    __shared__ int temp[THREADS_PER_BLOCK];
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    temp[threadIdx.x] = index < size ? a[index] * b[index] : 0;
    __syncthreads();
    if (threadIdx.x == 0) {
        int sum = 0;
        for (size_t i = 0; i < sizeof temp / sizeof(int); ++i) {
            sum += temp[i];
        }
        atomicAdd(result, sum);
    }
}

int main(void) {
    int a[N] = {22, 13, 16, 5};
    int b[N] = {5, 22, 17, 37};
    int answer;

    int *cuda_a, *cuda_b, *cuda_answer;
    hipMalloc(&cuda_a, sizeof a);
    hipMalloc(&cuda_b, sizeof b);
    hipMalloc(&cuda_answer, sizeof answer);
    hipMemcpy(cuda_a, a, sizeof a, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, sizeof b, hipMemcpyHostToDevice);

    vector_dot_product_cuda<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK,
                              THREADS_PER_BLOCK>>>(cuda_answer, cuda_a, cuda_b,
                                                   sizeof a / sizeof(int));
    hipDeviceSynchronize();

    hipMemcpy(&answer, cuda_answer, sizeof answer, hipMemcpyDeviceToHost);
    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_answer);

    print_vector("A", a, sizeof a / sizeof(int));
    print_vector("B", b, sizeof b / sizeof(int));
    printf("Answer = %d\n", answer);

    return 0;
}
