
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKS_COUNT 2
#define THREADS_PER_BLOCK 4

__host__ void hello_cpu() {
    printf("Hello from CPU!\n");
}

__global__ void hello_gpu() {
    printf("Hello from GPU%u[%u]!\n", blockIdx.x, threadIdx.x);
}

int main(void) {
    hello_cpu();
    hello_gpu<<<BLOCKS_COUNT, THREADS_PER_BLOCK>>>();
    hipDeviceSynchronize();
    return 0;
}
