
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define THREADS_PER_BLOCK 256

__global__ void aggregate(unsigned int items_count, unsigned int days_count,
                          const float prices[], unsigned int *const records[],
                          float total[]) {
    __shared__ float temp[THREADS_PER_BLOCK];
    unsigned int day = blockIdx.x;
    unsigned int index = threadIdx.x;
    unsigned int stride = THREADS_PER_BLOCK;
    temp[index] = 0;
    for (unsigned int i = index; i < items_count; i += stride) {
        temp[i] += prices[i] * records[i][day];
    }
    __syncthreads();
    if (index == 0) {
        float sum = 0;
        for (size_t i = 0; i < sizeof temp / sizeof(float); ++i) {
            sum += temp[i];
        }
        total[day] = sum;
    }
}

int main(int argc, char **argv) {
    if (argc != 2) {
        fprintf(stderr, "Usage: %s <path-to-records>\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    const char *records_filename = argv[1];
    FILE *records_file = fopen(records_filename, "r");
    if (records_file == NULL) {
        fprintf(stderr, "Failed to open file %s.\n", records_filename);
        exit(EXIT_FAILURE);
    }

    unsigned int items_count, days_count;
    fscanf(records_file, "%u%u", &items_count, &days_count);
    float *prices = (float *)malloc(items_count * sizeof(float));
    unsigned int **records =
        (unsigned int **)malloc(items_count * sizeof(unsigned int *));
    for (unsigned int i = 0; i < items_count; ++i) {
        fscanf(records_file, "%f", &prices[i]);
        records[i] = (unsigned int *)malloc(days_count * sizeof(unsigned int));
    }
    for (unsigned int i = 0; i < items_count; ++i) {
        for (unsigned int j = 0; j < days_count; ++j) {
            fscanf(records_file, "%u", &records[i][j]);
        }
    }
    fclose(records_file);

    float *prices_cuda;
    hipMalloc(&prices_cuda, items_count * sizeof(float));
    hipMemcpy(prices_cuda, prices, items_count * sizeof(float),
               hipMemcpyHostToDevice);
    unsigned int **records_cuda;
    hipMalloc(&records_cuda, items_count * sizeof(unsigned int *));
    unsigned int **records_cuda_ptrs =
        (unsigned int **)malloc(items_count * sizeof(unsigned int *));
    for (unsigned int i = 0; i < items_count; ++i) {
        hipMalloc(&records_cuda_ptrs[i], days_count * sizeof(unsigned int));
        hipMemcpy(records_cuda_ptrs[i], records[i],
                   days_count * sizeof(unsigned int), hipMemcpyHostToDevice);
    }
    hipMemcpy(records_cuda, records_cuda_ptrs,
               items_count * sizeof(unsigned int *), hipMemcpyHostToDevice);
    float *total_cuda;
    hipMalloc(&total_cuda, days_count * sizeof(float));

    aggregate<<<days_count, THREADS_PER_BLOCK>>>(
        items_count, days_count, prices_cuda, records_cuda, total_cuda);
    hipDeviceSynchronize();

    float *total = (float *)malloc(days_count * sizeof(float));
    hipMemcpy(total, total_cuda, days_count * sizeof(float),
               hipMemcpyDeviceToHost);
    hipFree(total_cuda);
    for (unsigned int i = 0; i < items_count; ++i) {
        hipFree(records_cuda_ptrs[i]);
    }
    free(records_cuda_ptrs);
    hipFree(records_cuda);
    hipFree(prices_cuda);

    for (unsigned int i = 0; i < days_count; ++i) {
        printf("%.2f ", total[i]);
    }
    printf("\n");

    free(total);
    for (unsigned int i = 0; i < items_count; ++i) {
        free(records[i]);
    }
    free(records);
    free(prices);

    return 0;
}
